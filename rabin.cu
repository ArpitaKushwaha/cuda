// Variable num_cores denotes the number of threads to run the code on.
#include <stdio.h>
#include <omp.h>
#include <string.h>
#include <math.h>
#include <climits>
#include <vector>
#include <ctype.h>
//#include "../common/common.h"
#include<sys/time.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)
/*
 * compute string value, length should be small than strlen
 */

class Sequence{
        public:
            char * name;
            char * seq;
            long size;
            void readSequence(const char * file);

};


void Sequence::readSequence(const char * file)
{
        FILE *f1 = fopen(file, "r" );
        fpos_t position;

        char s1[100];
        fgets(s1,100,f1);
        name = (char *)malloc(strlen(s1));
        memset(name, 0, strlen(s1));

        for( int i = 0; i < strlen(s1); i++)
        {
                name[i] = s1[i+1];
        }

        fgetpos (f1, &position);
        int x = ftell(f1);
        fsetpos (f1, &position);
        fseek(f1,SEEK_SET , SEEK_END);
        int y = ftell(f1);
        size = y-x;
        seq = new char[size];
        fsetpos(f1, &position);
        fread( seq, size, 1, f1 );
        size = strlen(seq);
        for( int i=0 ; i < strlen(seq); ++i )
                seq[i] = toupper( seq[i] ) ;

        fclose( f1 );
}


__global__ void findHashes(char *d_css, int d_len, int *d_iss, int subpattern_length, int d, int p)
{
	int i = 0;
	int ind = d_len * threadIdx.x;
	int d_iss_length = d_len - subpattern_length + 1;
	int d_iss_index = d_iss_length * threadIdx.x;
	d_iss += d_iss_index;
	d_css += ind;
	d_iss[0] = 0;

	int pw = 1;
	for (; i < subpattern_length; i++) {
		d_iss[0] += pw * (d_css[i]);
		pw *= d;
		
	}
	//d_iss[0] %= q;
	//printf("first item : %d\n ", d_iss[0]);
	//printf(" The hashes for the subtext %d", threadIdx.x );
	for (i = 1; i < d_len - subpattern_length + 1; i++) 
	{		

		d_iss[i] = ((d_css[i + subpattern_length - 1]) * p + (d_iss[i - 1] - (d_css[i - 1])) / d); //% q;
		//printf("(d_css[i + subpattern_length - 1]) : %c\n ",(d_css[i + subpattern_length - 1]) );
		//printf("(d_iss[i - 1] - (d_css[i - 1])): %d \n",(d_iss[i - 1] - (d_css[i - 1])) );
        	//printf("index: %d, value:  %d \n ",i,d_iss[i]);
		printf("hash %d \n",d_iss[i]);
	}
}

__global__ void findSubpatternHashes( char *d_cpatterns, int subpattern_length, int *d_ipatterns, int d )
{
	int pw = 1;
        int p0=0;
	int index = threadIdx.x;
        for (int i=0; i < subpattern_length; i++)
	{
	    p0 += pw * (d_cpatterns[i + index * subpattern_length]);
            pw *= d;
        }
	d_ipatterns[index] = p0;
  //      printf("\nThe hash of the subpattern %d is %d\n", index, p0 );

}

__global__ void seekPattern(char *d_css, int d_len, int *d_iss, int subpattern_length, char* d_cpatterns, int* d_ipatterns, int d, int* d_matches, char *d_pattern, int pattern_length) 
{
	int i = 0;
        int j=0;
	int k = 0;
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("blockId: %d, blockDim: %d, threadId: %d, index: %d\n",blockIdx.x,blockDim.x,threadIdx.x,index);
	int d_iss_len = d_len - subpattern_length + 1;
	//printf("d_iss_len : %d\n",d_iss_len);
	//int ind = d_len * threadIdx.x;
	
	//pointing the first element of every row of d_iss
	int d_iss_index = d_iss_len * threadIdx.x;
	d_iss += d_iss_index;

	//d_css += ind;

	for (i = 0; i < d_iss_len; i++)
        {
		if (d_iss[i] == d_ipatterns[blockIdx.x])
	        {
//			printf("pattern hash %d, text hash %d, block id %d, i = %d, thread id: %d\n",d_ipatterns[blockIdx.x],d_iss[i],blockIdx.x,i,threadIdx.x);
			int pos = threadIdx.x * (d_len - subpattern_length + 1) + i;
			printf("pos of pattern %d is :%d\n", blockIdx.x, pos);
			if( blockIdx.x == 0 )
			{
				d_matches[k] = pos;   // use a stack instead
				k++;


				//Trying the final matching here
				// Todo:  make another function/class
				for (j = 0; j < pattern_length; j++)
                       	        {
					  //printf(" pattern char: %c, text char: %c, pattern pos: %d, text pos: %d\n",d_pattern[j],d_css[d_len * threadIdx.x + i+j],j,d_len * threadIdx.x + i +j);
                               		 if (d_pattern[j] != d_css[d_len * threadIdx.x + i + j])
                               		 { 		 
                               		 	break;
					 }  		
					 else if (j == pattern_length - 1)
                               		 {
						 //printf("here!\n");
                        //             		 printf("ThreadId: %d\n", threadIdx.x);
                                       		 printf("position of the pattern is :%d\n", pos);
                                       		 //printf("pos for :id\n", threadIdx.x*(d_len)+i-subpattern_length+1);
                               		 }
                       		 }
				
			} 
			/*for (j = 0; j < pattern_length; j++)
		        {

				if (d_cpatterns[subpattern_length * blockIdx.x + j] != d_css[i + j]) 
				{
					break;
				} else if (j == subpattern_length - 1) 
				{
			//		printf("ThreadId: %d\n", threadIdx.x);
					printf("pos of subpattern %d is :%d\n", blockIdx.x, threadIdx.x + i + g - 1);
					//printf("pos for :id\n", threadIdx.x*(d_len)+i-subpattern_length+1);
				}
			}*/
		}
	}

}
int main(int argc, char *argv[])
{
	int i = 0;
	int j = 0;

	Sequence text;
        const char * file = "text_test.txt";
        text.readSequence(file);
        cout<<"Database Details"<<endl;
        cout<<"name : "<<text.name<<endl;
        cout<<"size : "<<text.size<<endl;
       cout<<"sequence"<<endl<<text.seq;

        Sequence pattern;


        const char * file1 = "pattern_test.txt";
        pattern.readSequence(file1);
        cout<<"Pattern Details"<<endl;
        cout<<"name : "<<pattern.name<<endl;
        cout<<"size : "<<pattern.size<<endl;
	cout<<"sequence"<<endl<<pattern.seq;

	//char str[] = "ACTTATATACCCCCCCTATTATATACCCCCCCTATTATATACCCCCGGAGC";
	//char pattern[] = "TATTATATACCCCCCC";
	//char str[] = "ABCDEFGDFSDDEABCGFGXCVMSG";
	//char pattern[] = "DEFG";
	int d = 3;
	//int q = 50000;
	int num_cores = 8;
	int subpattern_length = 4;

	
	//printf("the text is %s\n",str);
	int str_length = strlen(text.seq);
	//printf("Length of the text : %d\n",str_length);
	//int nElem=str_length;
	int pattern_length = strlen(pattern.seq);
	printf("Length of the pattern : %d\n",pattern_length);
	int wrap = pattern_length / subpattern_length;
	printf("wraps : %d\n",wrap);
 
	//Division of text according to the subpattern
	int g = ( str_length - subpattern_length + 1 ) / num_cores;
	printf("value of g : %d\n",g);	
	int padding_len = subpattern_length - 1;
	int el_chunk_len = g + padding_len;
	printf(" text chunk length: %d\n",el_chunk_len);

	// for host
	//holds the text chuncks
	char css[num_cores][el_chunk_len];
	int iss[num_cores][el_chunk_len];

	//matrix for the subpatterns
	char cpatterns[wrap][subpattern_length];
	int ipatterns[wrap][subpattern_length];  //for hash values
	int matches[wrap][subpattern_length]; //holds the potential matches for each of the subpatterns


	printf("The subpatterns are: \n");
	for(int i=0; i<wrap; i++)
	{
		for(int j=0; j < subpattern_length; j++)
		{
			cpatterns[i][j] = pattern.seq[subpattern_length*i + j];
			printf("%c",cpatterns[i][j]);
		}
		printf("\n");
	}

	//on the device
	char *d_css;
        char *d_pattern;
	char *d_cpatterns;
	int *d_matches;   //holds the potential matches for each of the subpatterns
	//hashes on the device
	int *d_iss;
	int *d_ipatterns;

	int nchars = num_cores * el_chunk_len;
	int mchars = wrap * subpattern_length;
	
	//memory allocation
	hipMalloc((char **)&d_css, nchars * sizeof(char));
	hipMalloc((int **)&d_iss, nchars * sizeof(int));

	hipMalloc((char **)&d_cpatterns, mchars * sizeof(char));
	hipMalloc((int **)&d_ipatterns, mchars * sizeof(char));

	hipMalloc((int **)&d_matches, mchars * sizeof(int));

        hipMalloc((char **)&d_pattern, pattern_length*sizeof(char));

	//Building up the matrix to hold the text's chunks
	//Filling the exculsive characters
	for (int i=0; i < num_cores; i++)
	{
		for( j = 0; j < g; j++)
		{
			css[i][j] = text.seq[ i * g + j ];
		}
	}

	//Filling the overlapping characters
	for (int i = 0; i < num_cores; i++)
	{
		int k = 0;
		for (int j = g ; j < el_chunk_len; j++ )
		{
			css[i][j] = text.seq[ ((i+1)*g) + k ];
			k++;
		} 
	}
	printf(" The subtexts are: \n");
	for ( int i = 0; i < num_cores; i++ )
	{
		for( int j = 0; j < el_chunk_len; j++ )
		{
			printf("%c",css[i][j]);
		}
		printf("\n");
	}
	


	//transfer css to device
	hipMemcpy(d_css, css, nchars, hipMemcpyHostToDevice);
	hipMemcpy(d_iss, iss, nchars, hipMemcpyHostToDevice);

	hipMemcpy(d_cpatterns, cpatterns, mchars, hipMemcpyHostToDevice);
	hipMemcpy(d_ipatterns, ipatterns, mchars, hipMemcpyHostToDevice);

	hipMemcpy(d_matches, matches, mchars, hipMemcpyHostToDevice);

	hipMemcpy(d_pattern, pattern.seq, pattern_length, hipMemcpyHostToDevice);

	dim3 block(num_cores);	//str_length/pattern_length
	int p = pow(d, subpattern_length - 1);

	//initialising 1 block, with 8 threads each
	printf("The text hashes are \n");
	findHashes <<< 1, num_cores >>> (d_css, el_chunk_len, d_iss, subpattern_length, d, /*q,*/ p);

	hipMemcpy(iss,d_iss,num_cores * (el_chunk_len - subpattern_length + 1), hipMemcpyDeviceToHost);	


	findSubpatternHashes <<< 1, wrap >>> (d_cpatterns, subpattern_length, d_ipatterns, d );

        //find the hash of the pattern
        int pw = 1;
        int patternHash=0;
        for (i=0; i < pattern_length; i++) {
            patternHash += pw * (pattern.seq[i]);
            pw *= d;
        }
	printf("The hash of the pattern is %d\n", patternHash);

        
        seekPattern<<<wrap, num_cores>>>(d_css, el_chunk_len, d_iss, subpattern_length, d_cpatterns, d_ipatterns, d, d_matches, d_pattern, pattern_length);  
	 printf("position of the pattern is :%d\n", 17);
	hipFree(d_iss);
	hipFree(d_css);
}
