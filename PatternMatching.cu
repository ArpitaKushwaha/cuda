// Variable num_cores denotes the number of threads to run the code on.

#include <stdio.h>
//#include <omp.h>
#include <string.h>
#include <math.h>
//#include "../common/common.h"
#include <hip/hip_runtime.h>

/*
 * compute string value, length should be small than strlen
 */

__global__ void findHashes(char *d_css, int d_len, int *d_iss, int subpattern_length, int d, int p)
{
	int i = 0;
	int ind = d_len * threadIdx.x;
	int d_iss_length = d_len - subpattern_length + 1;
	int d_iss_index = d_iss_length * threadIdx.x;
	d_iss += d_iss_index;
	d_css += ind;
	d_iss[0] = 0;

	int pw = 1;
	for (; i < subpattern_length; i++) {
		d_iss[0] += pw * (d_css[i]);
		pw *= d;
		
	}
	//d_iss[0] %= q;
	//printf("first item : %d\n ", d_iss[0]);
	//printf(" The hashes for the subtext %d", threadIdx.x );
	for (i = 1; i < d_len - subpattern_length + 1; i++) 
	{		

		d_iss[i] = ((d_css[i + subpattern_length - 1]) * p + (d_iss[i - 1] - (d_css[i - 1])) / d); //% q;
		//printf("(d_css[i + subpattern_length - 1]) : %c\n ",(d_css[i + subpattern_length - 1]) );
		//printf("(d_iss[i - 1] - (d_css[i - 1])): %d \n",(d_iss[i - 1] - (d_css[i - 1])) );
        	//printf("index: %d, value:  %d \n ",i,d_iss[i]);
	}
}

__global__ void findSubpatternHashes( char *d_cpatterns, int subpattern_length, int *d_ipatterns, int d )
{
	int pw = 1;
        int p0=0;
	int index = threadIdx.x;
        for (int i=0; i < subpattern_length; i++)
	{
	    p0 += pw * (d_cpatterns[i + index * subpattern_length]);
            pw *= d;
        }
	d_ipatterns[index] = p0;
        printf("\nThe hash of the subpattern %d is %d\n", index, p0 );

}

__global__ void seekPattern(char *d_css, int d_len, int *d_iss, int subpattern_length, char* d_cpatterns, int* d_ipatterns, int d, int* d_matches, char *d_pattern, int pattern_length) 
{
	int i = 0;
        int j=0;
	int k = 0;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("blockId: %d, blockDim: %d, threadId: %d, index: %d\n",blockIdx.x,blockDim.x,threadIdx.x,index);
	int d_iss_len = d_len - subpattern_length + 1;
	//printf("d_iss_len : %d\n",d_iss_len);
	int ind = d_len * threadIdx.x;
	
	//pointing the first element of every row of d_iss
	int d_iss_index = d_iss_len * threadIdx.x;
	d_iss += d_iss_index;

	//d_css += ind;

	for (i = 0; i < d_iss_len; i++)
        {
		if (d_iss[i] == d_ipatterns[blockIdx.x])
	        {
			printf("pattern hash %d, text hash %d, block id %d, i = %d, thread id: %d\n",d_ipatterns[blockIdx.x],d_iss[i],blockIdx.x,i,threadIdx.x);
			int pos = threadIdx.x * (d_len - subpattern_length + 1) + i;
			printf("pos of subpattern %d is :%d\n", blockIdx.x, pos);
			if( blockIdx.x == 0 )
			{
				d_matches[k] = pos;   // use a stack instead
				k++;


				//Trying the final matching here
				// Todo:  make another function/class
				for (j = 0; j < pattern_length; j++)
                       	        {
					  printf(" pattern char: %c, text char: %c, pattern pos: %d, text pos: %d\n",d_pattern[j],d_css[d_len * threadIdx.x + i+j],j,d_len * threadIdx.x + i +j);
                               		 if (d_pattern[j] != d_css[d_len * threadIdx.x + i + j])
                               		 { 		 
                               		 	break;
					 }  		
					 else if (j == pattern_length - 1)
                               		 {
						 //printf("here!\n");
                        //             		 printf("ThreadId: %d\n", threadIdx.x);
                                       		 printf("position of the pattern is :%d\n", pos);
                                       		 //printf("pos for :id\n", threadIdx.x*(d_len)+i-subpattern_length+1);
                               		 }
                       		 }
				
			} 
			/*for (j = 0; j < pattern_length; j++)
		        {

				if (d_cpatterns[subpattern_length * blockIdx.x + j] != d_css[i + j]) 
				{
					break;
				} else if (j == subpattern_length - 1) 
				{
			//		printf("ThreadId: %d\n", threadIdx.x);
					printf("pos of subpattern %d is :%d\n", blockIdx.x, threadIdx.x + i + g - 1);
					//printf("pos for :id\n", threadIdx.x*(d_len)+i-subpattern_length+1);
				}
			}*/
		}
	}

}
int main(int argc, char *argv[])
{
	int i = 0;
	int j = 0;
	//char str[] = "ACTTATATACCCCCCCTATTATATACCCCCCCTATTATATACCCCCGGAGC";
	//char pattern[] = "TATTATATACCCCCCC";
	char str[] = "ABCDEFGDFSDDEABCGFGXCVMSG";
	char pattern[] = "DEFG";
	int d = 3;
	//int q = 50000;
	int num_cores = 8;
	int subpattern_length = 4;

	
	printf("the text is %s\n",str);
	int str_length = strlen(str);
	printf("Length of the text : %d\n",str_length);
	//int nElem=str_length;
	int pattern_length = strlen(pattern);
	printf("Length of the pattern : %d\n",pattern_length);
	int wrap = pattern_length / subpattern_length;
	printf("wraps : %d\n",wrap);
 
	//Division of text according to the subpattern
	int g = ( str_length - subpattern_length + 1 ) / num_cores;
	printf("value of g : %d\n",g);	
	int padding_len = subpattern_length - 1;
	int el_chunk_len = g + padding_len;
	printf(" text chunk length: %d\n",el_chunk_len);

	// for host
	//holds the text chuncks
	char css[num_cores][el_chunk_len];
	int iss[num_cores][el_chunk_len];

	//matrix for the subpatterns
	char cpatterns[wrap][subpattern_length];
	int ipatterns[wrap][subpattern_length];  //for hash values
	int matches[wrap][subpattern_length]; //holds the potential matches for each of the subpatterns


	printf("The subpatterns are: \n");
	for(int i=0; i<wrap; i++)
	{
		for(int j=0; j < subpattern_length; j++)
		{
			cpatterns[i][j] = pattern[subpattern_length*i + j];
			printf("%c",cpatterns[i][j]);
		}
		printf("\n");
	}

	//on the device
	char *d_css;
        char *d_pattern;
	char *d_cpatterns;
	int *d_matches;   //holds the potential matches for each of the subpatterns
	//hashes on the device
	int *d_iss;
	int *d_ipatterns;

	int nchars = num_cores * el_chunk_len;
	int mchars = wrap * subpattern_length;
	
	//memory allocation
	hipMalloc((char **)&d_css, nchars * sizeof(char));
	hipMalloc((int **)&d_iss, nchars * sizeof(int));

	hipMalloc((char **)&d_cpatterns, mchars * sizeof(char));
	hipMalloc((int **)&d_ipatterns, mchars * sizeof(char));

	hipMalloc((int **)&d_matches, mchars * sizeof(int));

        hipMalloc((char **)&d_pattern, pattern_length*sizeof(char));

	//Building up the matrix to hold the text's chunks
	//Filling the exculsive characters
	for (int i=0; i < num_cores; i++)
	{
		for( j = 0; j < g; j++)
		{
			css[i][j] = str[ i * g + j ];
		}
	}

	//Filling the overlapping characters
	for (int i = 0; i < num_cores; i++)
	{
		int k = 0;
		for (int j = g ; j < el_chunk_len; j++ )
		{
			css[i][j] = str[ ((i+1)*g) + k ];
			k++;
		} 
	}
	printf(" The subtexts are: \n");
	for ( int i = 0; i < num_cores; i++ )
	{
		for( int j = 0; j < el_chunk_len; j++ )
		{
			printf("%c",css[i][j]);
		}
		printf("\n");
	}
	


	//transfer css to device
	hipMemcpy(d_css, css, nchars, hipMemcpyHostToDevice);
	hipMemcpy(d_iss, iss, nchars, hipMemcpyHostToDevice);

	hipMemcpy(d_cpatterns, cpatterns, mchars, hipMemcpyHostToDevice);
	hipMemcpy(d_ipatterns, ipatterns, mchars, hipMemcpyHostToDevice);

	hipMemcpy(d_matches, matches, mchars, hipMemcpyHostToDevice);

	hipMemcpy(d_pattern, pattern, pattern_length, hipMemcpyHostToDevice);

	dim3 block(num_cores);	//str_length/pattern_length
	//__global__ void findHashes(char *d_css, int d_len, int *d_iss, int pattern_length, int d, int q, int p)
	int p = pow(d, subpattern_length - 1);

	//initialising 1 block, with 8 threads each
	printf("The text hashes are \n");
	findHashes <<< 1, num_cores >>> (d_css, el_chunk_len, d_iss, subpattern_length, d, /*q,*/ p);

	hipMemcpy(iss,d_iss,num_cores * (el_chunk_len - subpattern_length + 1), hipMemcpyDeviceToHost);	

	/*for( int i =0; i < num_cores; i++)
	{
		for( int j=0; j < el_chunk_len - subpattern_length + 1; j++)
		{
			printf("%d  ",iss[i][j]);
		}
		printf("\n");
	}*/

	findSubpatternHashes <<< 1, wrap >>> (d_cpatterns, subpattern_length, d_ipatterns, d );

        //find the hash of the pattern
        int pw = 1;
        int patternHash=0;
        for (i=0; i < pattern_length; i++) {
            patternHash += pw * (pattern[i]);
            pw *= d;
        }
	printf("The hash of the pattern is %d\n", patternHash);

	//finding hashes of the subpatterns sequenctially
	/* printf("hashes calculated sequentially\n");
	for(int i = 0; i < wrap; i++)
	{
		int pw = 1;
		int p0 = 0;
		for( int j = 0; j<subpattern_length; j++)
		{
			p0 += pw * cpatterns[i][j];
			pw *= d;
		}
		printf("%d\n",p0);
	}*/
        
        seekPattern<<<wrap, num_cores>>>(d_css, el_chunk_len, d_iss, subpattern_length, d_cpatterns, d_ipatterns, d, d_matches, d_pattern, pattern_length);  

	//printf("%d %d %d %d %d \n", el_chunk_len, pattern_length, d, q, p);

	//cudaMemcpy(iss, d_iss, nchars * sizeof(int), cudaMemcpyDeviceToHost);
	/*for (i=0;i<num_cores;i++)
	   {
	   for (j=0;j<el_chunk_len;j++)
	   	printf("%d ", iss[i][j]);
	   printf("\n");
	   } 
	*/
	hipFree(d_iss);
	hipFree(d_css);
}
